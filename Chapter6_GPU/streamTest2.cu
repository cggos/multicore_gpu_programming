/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : January 2015
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc streamTest2.cu -o streamTest2
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

const int DATASIZE=1024;

__global__ void doSmt(int *data)
{
  // Simplification of above 
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  + 
               blockIdx.y * gridDim.x + 
               blockIdx.x ) * blockDim.x + 
               threadIdx.x; 

  printf ("Hello world from %i\n", myID);
}

int main ()
{
  hipStream_t str[2];
  int *h_data[2], *d_data[2];
  int i;

  for(i=0;i<2;i++)
  {
    hipHostMalloc((void **) &(h_data[i]), sizeof(int) * DATASIZE, hipHostMallocDefault);
    hipMalloc((void ** )&(d_data[i]), sizeof(int) * DATASIZE);
  }
    // inititalize h_data[i]....
  
  for(i=0;i<2;i++)
  {
    hipStreamCreate(&(str[i]));
    hipMemcpyAsync(d_data[i], h_data[i], sizeof(int) * DATASIZE, hipMemcpyHostToDevice, str[i]);
   
    doSmt <<< 10, 256, 0, str[i] >>> (d_data[i]);
    
    hipMemcpyAsync(h_data[i], d_data[i], sizeof(int) * DATASIZE, hipMemcpyDeviceToHost, str[i]);    
  }
    
  hipStreamSynchronize(str[0]);
  hipStreamSynchronize(str[1]);
  hipStreamDestroy(str[0]);
  hipStreamDestroy(str[1]);

  for(i=0;i<2;i++)
  {
    hipFree(h_data[i]);
    hipFree(d_data[i]);
  }
  hipDeviceReset();
  return 1;
}
