/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.01
 Last modified : February 2015
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc memcpyTest.cu -o memcpyTest
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

const int MAXDATASIZE = 1024 * 1024;

int main (int argc, char **argv)
{
  int iter = atoi (argv[1]);
  int step = atoi (argv[2]);
  hipStream_t str;
  int *h_data, *d_data;
  int i, dataSize;;
  hipEvent_t startT, endT;
  float duration;

  hipHostMalloc ((void **) &h_data, sizeof (int) * MAXDATASIZE, hipHostMallocDefault);
  hipMalloc ((void **) &d_data, sizeof (int) * MAXDATASIZE);
  for (i = 0; i < MAXDATASIZE; i++)
    h_data[i] = i;

  hipEventCreate (&startT);
  hipEventCreate (&endT);
  hipStreamCreate (&str);

  for (dataSize = 0; dataSize <= MAXDATASIZE; dataSize += step)
    {
      hipEventRecord (startT, str);
      for (i = 0; i < iter; i++)
        {
          hipMemcpyAsync (d_data, h_data, sizeof (int) * dataSize, hipMemcpyHostToDevice, str);
        }
      hipEventRecord (endT, str);
      hipEventSynchronize (endT);
      hipEventElapsedTime (&duration, startT, endT);
      printf ("%i %f\n", (int) (dataSize * sizeof (int)), duration / iter);
    }

  hipStreamDestroy (str);
  hipEventDestroy (startT);
  hipEventDestroy (endT);

  hipHostFree (h_data);
  hipFree (d_data);
  hipDeviceReset ();
  return 1;
}
