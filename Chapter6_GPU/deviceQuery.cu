/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc deviceQuery.cu -o deviceQuery
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

int main ()
{
  int deviceCount = 0;
  hipGetDeviceCount (&deviceCount);
  if (deviceCount == 0)
    printf ("No CUDA compatible GPU.\n");
  else
    {
      hipDeviceProp_t pr;
      for (int i = 0; i < deviceCount; i++)
        {
          hipGetDeviceProperties (&pr, i);
          printf ("Dev #%i is %s\n", i, pr.name);
        }
    }
  return 1;
}
