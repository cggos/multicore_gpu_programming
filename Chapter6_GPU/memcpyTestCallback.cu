/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc memcpyTestCallback.cu -o memcpyTestCallback
 ============================================================================
 */
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

const int MAXDATASIZE = 1024 * 1024;
//---------------------------------------------------------
void myCallBack (hipStream_t stream, hipError_t status, void *userData)
{
  float *t = (float *) userData;
  clock_t x = clock();
  *t = x*1.0/CLOCKS_PER_SEC;
}
//---------------------------------------------------------
int main (int argc, char **argv)
{
  int iter = atoi (argv[1]);
  int step = atoi (argv[2]);
  hipStream_t str;
  int *h_data, *d_data;
  int i, dataSize;;

  hipStreamCreate(&str);
  hipHostMalloc ((void **) &h_data, sizeof (int) * MAXDATASIZE, hipHostMallocDefault);
  hipMalloc ((void **) &d_data, sizeof (int) * MAXDATASIZE);
  for (i = 0; i < MAXDATASIZE; i++)
    h_data[i] = i;

  float t1, t2;
  hipStreamAddCallback (str, myCallBack, (void *) &t1, 0);
  for (dataSize = 0; dataSize <= MAXDATASIZE; dataSize += step)
    {
      for (i = 0; i < iter; i++)
        {
          hipMemcpyAsync (d_data, h_data, sizeof (int) * dataSize, hipMemcpyHostToDevice, str);
        }
      hipStreamAddCallback (str, myCallBack, (void *) &t2, 0);
      hipStreamSynchronize(str);
      printf ("%i %f\n", (int) (dataSize * sizeof (int)), (t2 - t1) / iter);
      t1 = t2;
    }

  hipStreamDestroy (str);

  hipHostFree (h_data);
  hipFree (d_data);
  hipDeviceReset ();
  return 1;
}
