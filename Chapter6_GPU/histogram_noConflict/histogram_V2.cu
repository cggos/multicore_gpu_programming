#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : Maximum number of bins are used
                 warpSize is assumed to be fixed to 32
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "../common/pgm.h"

const int BINS = 256;
const int BLOCKSIZE = 192;
const int MAXPIXELSPERTHREAD = 255; // to avoid overflowing a byte counter
const int BINS4ALL = BINS * BLOCKSIZE;

//*****************************************************************
void CPU_histogram (unsigned char *in, int N, int *h, int bins)
{
  int i;
  // initialize histogram counts
  for (i = 0; i < bins; i++)
    h[i] = 0;

  // accummulate counts
  for (i = 0; i < N; i++)
    h[in[i]]++;
}

//*****************************************************************
__global__ void GPU_histogram_V2 (int *in, int N, int *h)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  int locID = threadIdx.x;
  int GRIDSIZE = gridDim.x * blockDim.x;
  __shared__ unsigned char localH[BINS4ALL];
  int bankID = locID;
  int i;

  // initialize the local, shared-memory bins
  for (i = locID; i < BINS4ALL; i += blockDim.x)
    localH[i] = 0;

  // wait for all warps to complete the previous step
  __syncthreads ();

  //start processing the image data
  unsigned char *mySharedBank = localH + bankID;
  for (i = gloID; i < N; i += GRIDSIZE)
      {
        int temp = in[i];
        int v = temp & 0xFF;
        int v2 = (temp >> 8) & 0xFF;
        int v3 = (temp >> 16) & 0xFF;
        int v4 = (temp >> 24) & 0xFF;
        mySharedBank[v * BLOCKSIZE]++;  
        mySharedBank[v2 * BLOCKSIZE]++;
        mySharedBank[v3 * BLOCKSIZE]++;
        mySharedBank[v4 * BLOCKSIZE]++;
      }

  // wait for all warps to complete the local calculations, before updating the global counts
  __syncthreads ();

  // use atomic operations to add the local findings to the global memory bins 
  for (i = locID; i < BINS4ALL; i += blockDim.x)
    atomicAdd (h + (i/BLOCKSIZE), localH[i]); 
}

//*****************************************************************
int main (int argc, char **argv)
{

  PGMImage inImg (argv[1]);

  int *d_in, *h_in;
  int *d_hist, *h_hist, *cpu_hist;
  int i, N, bins;

  h_in = (int *) inImg.pixels;
  N = ceil ((inImg.x_dim * inImg.y_dim) / 4.0);

  bins = inImg.num_colors + 1;
  h_hist = (int *) malloc (bins * sizeof (int));
  cpu_hist = (int *) malloc (bins * sizeof (int));

  CPU_histogram (inImg.pixels, inImg.x_dim * inImg.y_dim, cpu_hist, bins);

  // ensures that shared memory/L1 cache are split in 48KB/16KB configuration
  hipDeviceSetCacheConfig( hipFuncCachePreferShared);

  // timing related definitions  
  hipStream_t str;
  hipEvent_t startT, endT;
  float duration;

  // initialize two events
  hipStreamCreate (&str);
  hipEventCreate (&startT);
  hipEventCreate (&endT);

  hipMalloc ((void **) &d_in, sizeof (int) * N);
  hipMalloc ((void **) &d_hist, sizeof (int) * bins);
  hipMemcpy (d_in, h_in, sizeof (int) * N, hipMemcpyHostToDevice);
  hipMemset (d_hist, 0, bins * sizeof (int));

  int blocks = (int)ceil(N*4.0/(BLOCKSIZE * MAXPIXELSPERTHREAD));
    
  hipEventRecord (startT, str);
  GPU_histogram_V2 <<< blocks, BLOCKSIZE, 0, str >>> (d_in, N, d_hist);
  hipEventRecord (endT, str);

  // wait for endT event to take place
  hipEventSynchronize (endT);

  hipMemcpy (h_hist, d_hist, sizeof (int) * bins, hipMemcpyDeviceToHost);

  for (i = 0; i < BINS; i++)
    printf ("%i %i %i\n", i, cpu_hist[i], h_hist[i]);

  for (i = 0; i < BINS; i++)
    if (cpu_hist[i] != h_hist[i])
      printf ("Calculation mismatch (static) at : %i\n", i);

// calculate elapsed time
  hipEventElapsedTime (&duration, startT, endT);
  printf ("Kernel executed for %f ms\n", duration);

// clean-up allocated objects and reset device
  hipStreamDestroy (str);
  hipEventDestroy (startT);
  hipEventDestroy (endT);

  hipFree ((void *) d_in);
  hipFree ((void *) d_hist);
  free (h_hist);
  free (cpu_hist);
  hipDeviceReset ();

  return 0;
}
