#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.01
 Last modified : January 2015
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc odd.cu -o odd
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAXVALUE 10000

//------------------------------------
void numberGen (int N, int max, int *store)
{
  int i;
  srand (time (0));
  for (i = 0; i < N; i++)
    store[i] = rand () % max;
}

//------------------------------------

__global__ void countOdds (int *d, int N, int *odds)
{
  extern __shared__ int count[];

  int myID = blockIdx.x * blockDim.x + threadIdx.x;
  int localID = threadIdx.x;
  count[localID] = 0;
  if (myID < N)
    count[localID] = (d[myID] % 2);
  __syncthreads ();

  // reduction phase: sum up the block
  int step = 1;
  while (((localID | step) < blockDim.x) && ((localID & step) == 0))
    {
      count[localID] += count[localID | step];
      step <<= 1;
      __syncthreads ();
    }

  // slightly faster reduction code:  
//   int otherIdx = localID | step;  
//   while ((otherIdx < blockDim.x) && ((localID & step) == 0) )
//     {
//       count[localID] += count[otherIdx];
//       step <<= 1;
//       otherIdx = localID | step;  
//       __syncthreads ();
//     }
    
  // add to global counter
  if (localID == 0)
    atomicAdd (odds, count[0]);
}

//------------------------------------
int sharedSize (int b)
{
  return b * sizeof (int);
}

//------------------------------------

int main (int argc, char **argv)
{
  int N = atoi (argv[1]);

  int *ha, *hres, *da, *dres;   // host (h*) and device (d*) pointers

  ha = new int[N];
  hres = new int[1];

  hipMalloc ((void **) &da, sizeof (int) * N);
  hipMalloc ((void **) &dres, sizeof (int) * 1);

  numberGen (N, MAXVALUE, ha);

  hipMemcpy (da, ha, sizeof (int) * N, hipMemcpyHostToDevice);
  hipMemset (dres, 0, sizeof (int));

  int blockSize, gridSize;
  hipOccupancyMaxPotentialBlockSizeVariableSMem (&gridSize, &blockSize, (void *) countOdds, sharedSize, N);

  gridSize = ceil (1.0 * N / blockSize);
  printf ("Grid : %i    Block : %i\n", gridSize, blockSize);
  countOdds <<< gridSize, blockSize, blockSize * sizeof (int) >>> (da, N, dres);

  hipMemcpy (hres, dres, sizeof (int), hipMemcpyDeviceToHost);

  // correctness check
  int oc = 0;
  for (int i = 0; i < N; i++)
    if (ha[i] % 2)
      oc++;

  printf ("%i %i\n", *hres, oc);

  hipFree ((void *) da);
  hipFree ((void *) dres);
  delete[]ha;
  delete[]hres;
  hipDeviceReset ();

  return 0;
}
